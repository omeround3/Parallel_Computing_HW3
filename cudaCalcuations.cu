#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include "sequence_alignment.h"

__device__ int dev_strlen(char *dest) {
	int count = 0;
	char c = dest[count];
	while (c != '\0') {
		count++;
		c = dest[count];
	}
	return count;

}

__device__ char* dev_strcpy(char *dest, char *source) {
	char *ptr = dest;
	while (*source != '\0') {
		*dest = *source;
		dest++;
		source++;
	}
	*dest = '\0';
	return ptr;
}

__device__ Score* dev_copy(const Score *source, Score *dest) {
	dest->offset = source->offset;
	dest->hyphen_idx = source->hyphen_idx;
	dest->char_val = source->char_val;
	dest->alignment_score = source->alignment_score;
	dest->max_score = source->max_score;
	return dest;
}

__device__ Score* dev_compare(const Payload *d, Score *a, char *chars_comparision,
		double *weights) {
	int c1, c2;
	a->alignment_score = 0;
	for (int chr_ofst = 0; chr_ofst < d->len; ++chr_ofst) {
		c1 = d->seq1[chr_ofst + a->offset] - 'A';
		if (chr_ofst == a->hyphen_idx) {
			c2 = a->char_val;
		} else {
			c2 = d->seq2[chr_ofst] - 'A';
		}
		switch (chars_comparision[c1 * CHARS + c2]) {
		case '$':
			a->alignment_score += weights[0];
			break;
		case '%':
			a->alignment_score -= weights[1];
			break;
		case '#':
			a->alignment_score -= weights[2];
			break;
		default:
			a->alignment_score -= weights[3];
			break;
		}
	}
	return a;
}

__device__ Score* dev_compare_and_swap(const Score *a1, Score *a2) {
	if (a1->alignment_score > a2->alignment_score) {
		dev_copy(a1, a2);
	}
	return a2;
}
__device__ Score* dev_find_offset(const Payload *source, Score *res,
		char *chars_comparision, double *weights) {

	Score tmp;
	dev_copy(res, &tmp);

	for (int i = 1; i <= source->max_offset; ++i) {
		tmp.offset = i;
		dev_compare(source, &tmp, chars_comparision, weights);
		dev_compare_and_swap(&tmp, res);
	}
	return res;

}

__global__ void find_optimum(Payload *data, Score *results, char *chars_comparision,
		double *weights, int from) {
	Score tmp;

// Each thread will write to element idx
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

// Each block will be responsible to an idx in seq2
	int hyphen_idx = from + blockIdx.x;

// Each thread in block will replace to a different char (CHARS threads in total)
	int new_chr = threadIdx.x;

// "res_tmp" will hold the in/max element, "mut_tmp" is an helper
	dev_copy(&results[idx], &tmp);

// Set char to replace
	tmp.hyphen_idx = hyphen_idx;

// Set target char (if possible to replace)
	int c1 = data->seq2[hyphen_idx] - 'A';
	char sign = chars_comparision[c1 * CHARS + new_chr];
	if (sign != '%' && sign != '$') {
		tmp.char_val = new_chr;
		tmp.offset = 0;
		dev_compare(data, &tmp, chars_comparision, weights);
		dev_find_offset(data, &tmp, chars_comparision, weights);
		dev_compare_and_swap(&tmp, &results[idx]);
	}
}

void cuda_free(void *ptr, hipError_t err) {
// Free allocated memory on GPU - ArrayA
	if (hipFree(ptr) != hipSuccess) {
		fprintf(stderr, "Failed to free device data - %s\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void validate(hipError_t err) {
	if (err != hipSuccess) {
		fprintf(stderr, "%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

Score* computeOnGPU(Payload *data, Score *res, char *chars_comparision, double *weights, int from, int to) {
	omp_set_num_threads(THREADS);

	Score tmp;
	deep_copy_score(res, &tmp);

	int share = to - from;
	size_t size = sizeof(Score) * share * CHARS;
	size_t num_of_res = share * CHARS;

// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

// Allocate results array
	Score *results_array = (Score*) malloc(size);
#pragma omp parallel for
	for (int i = 0; i < num_of_res; ++i) {
		deep_copy_score(res, &results_array[i]);
	}
// Allocate memory on GPU for Source Score
	Payload *sourceGPU;
	err = hipMalloc((void**) &sourceGPU, sizeof(Payload));
	validate(err);

// Allocate memory on GPU for Results Alignments
	Score *resultsGPU;
	err = hipMalloc((void**) &resultsGPU, size);
	validate(err);

// Allocate memory on GPU for chars_comparision matrix
	char *pairsGPU;
	err = hipMalloc((void**) &pairsGPU, CHARS * CHARS);
	validate(err);

// Allocate memory on GPU for weights array
	double *weightsGPU;
	err = hipMalloc((void**) &weightsGPU, sizeof(double) * WEIGHTS_NUM);
	validate(err);

// Copy source from host to the GPU memory
	err = hipMemcpy(sourceGPU, data, sizeof(Payload), hipMemcpyHostToDevice);
	validate(err);

// Copy result from host to the GPU memory
	err = hipMemcpy(resultsGPU, results_array, size, hipMemcpyHostToDevice);
	validate(err);

// Copy chars_comparision matrix from host to the GPU memory
	err = hipMemcpy(pairsGPU, chars_comparision, CHARS * CHARS, hipMemcpyHostToDevice);
	validate(err);

// Copy weights array from host to the GPU memory
	err = hipMemcpy(weightsGPU, weights, sizeof(double) * WEIGHTS_NUM,
			hipMemcpyHostToDevice);
	validate(err);

// Launch the Kernel
	find_optimum<<<share, CHARS>>>(sourceGPU, resultsGPU, pairsGPU, weightsGPU,from);
	err = hipGetLastError();
	validate(err);
// Copy the  result from GPU to the host memory.
	err = hipMemcpy(results_array, resultsGPU, size, hipMemcpyDeviceToHost);
	validate(err);

// Find optimum in results using openmp
//	printf("SHARE: %ld\n", num_of_res);
#pragma omp parallel firstprivate(tmp) private(from,to,share)
	{
		int t_num = omp_get_thread_num();
		share = num_of_res / THREADS;
		from = t_num * share;
		to = t_num != THREADS - 1 ? (t_num + 1) * share : num_of_res;
		for (int i = from; i < to; ++i) {
			compare_scores_and_swap(&results_array[i], &tmp);
		}
#pragma omp critical
		{
			compare_scores_and_swap(&tmp, res);
		}
	}

// Free allocated space in GPU
	cuda_free(sourceGPU, err);
	cuda_free(resultsGPU, err);
	cuda_free(pairsGPU, err);
	cuda_free(weightsGPU, err);
	free(results_array);
	return res;
}

